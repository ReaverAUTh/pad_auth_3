#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include "supplementary.h"

//!------------------------------------------------------------------
//! EDIT THESE VALUES ACCORDING TO THE IMAGE SIZE AND SPECIFICATIONS

#define PIXELS 64 // PIXELS x PIXELS
#define PATCH_SIZE 3
#define FILTER_SIGMA 0.0185
#define PATCH_SIGMA 3.1550

//!------------------------------------------------------------------

// Device global variables
__device__ const int DEV_PIXELS = PIXELS;
__device__ const int DEV_PATCH_SIZE = PATCH_SIZE;
__device__ const float DEV_FILTER_SIGMA = (float)FILTER_SIGMA;
__device__ const int DEV_PADDING = PATCH_SIZE/2;

// Host global variables
const int HOST_PADDING = PATCH_SIZE/2;

// Functions
__global__ void denoise_image(float *filtered_image, float *image, int padded_size, float *G);
__device__ void compare_patches(float *comp_value, float *patch_i, int j, float *G, float *shared_memory);
__host__ float *nonLocalMeans(float *host_image);
__host__ float *gaussian_filter();



__host__ int main(){
	//read image from txt
    float *host_image = image_from_txt(PIXELS, HOST_PADDING);
	
	float *filtered_image;
	hipMallocManaged(&filtered_image,0*sizeof(float));
    struct timespec tic, toc;
	
    clock_gettime(CLOCK_MONOTONIC, &tic);
    filtered_image = nonLocalMeans(host_image);
    clock_gettime(CLOCK_MONOTONIC, &toc);
	
    FILE *f = fopen("filtered_image.txt", "w");
	if(f == NULL){
		printf("Cannot open filtered_image.txt\n");
		exit(1);
	}
	
    int pixels_counter = 0;
	int padded_size = PIXELS*PIXELS + 4*HOST_PADDING*PIXELS + 4*HOST_PADDING*HOST_PADDING;
	int start = PIXELS*HOST_PADDING + 2*HOST_PADDING*HOST_PADDING + HOST_PADDING; //skip first padding rows
    
	for(int i=start; i<(padded_size-start); i++){
        fprintf(f, "%f ", filtered_image[i]);
        pixels_counter++;
        if(pixels_counter == PIXELS){
            pixels_counter = 0;
			i += 2*HOST_PADDING;
            fprintf(f, "\n");
        }
    }
    fclose(f);
    free(host_image);
	hipFree(filtered_image);
	
	printf("*NLM-CUDA-SHARED Duration = %f second(s)* || (Pixels, Patch) = (%d, %d)\n", elapsed_time(tic,toc), PIXELS, PATCH_SIZE);
    return 0;
}

__host__ float *nonLocalMeans(float *host_image){
	int padded_size = PIXELS*PIXELS + 4*HOST_PADDING*PIXELS + 4*HOST_PADDING*HOST_PADDING;

    float *G;
	hipMallocManaged(&G, PATCH_SIZE*PATCH_SIZE*sizeof(float));
	if(G == NULL){
        exit(1);
    }
	float *temp = gaussian_filter();
	memcpy(G, temp, PATCH_SIZE*PATCH_SIZE*sizeof(float));
	
	//host_image is not know to both the host and device, hence the memcpy
	float *image;
	hipMallocManaged(&image, padded_size*sizeof(float));
	if(image == NULL){
        exit(1);
    }
	memcpy(image, host_image, padded_size*sizeof(float));

	float *filtered_image;
	hipMallocManaged(&filtered_image, padded_size*sizeof(float));
	if(filtered_image == NULL){
        exit(1);
    }
	// Fill array with -1, so after adding the image's values
	// the padding will have -1 values
	for(int i=0; i<padded_size; i++){
		filtered_image[i]=(float)-1;
	}
	
    //! KERNEL
	int shared_memory_size = PATCH_SIZE*(PIXELS + 2*HOST_PADDING);
    denoise_image<<<PIXELS, PIXELS, shared_memory_size*sizeof(float)>>>(filtered_image, image, padded_size, G);
	hipDeviceSynchronize();
	//! KERNEL
	
	hipFree(G);
	hipFree(image);
    return filtered_image;
}

//! Compute the gaussian filter
__host__ float *gaussian_filter(){
    float *G = (float *)malloc(PATCH_SIZE*PATCH_SIZE*sizeof(float));
    if(G == NULL){
        exit(1);
	}
	// bound for the 2D Gaussian filter
    int bound = PATCH_SIZE/2;
    for(int x=-bound; x<=bound; x++){
        for(int y=-bound; y<=bound; y++){
			int index = (x+bound)*PATCH_SIZE + (y+bound);
            G[index] = exp( -(float)(x*x+y*y)/(float)(2*PATCH_SIGMA*PATCH_SIGMA) ) / (float)(2*M_PI*PATCH_SIGMA*PATCH_SIGMA);
        }
    }
    return G;
}

__global__ void denoise_image(float *filtered_image, float *image, int padded_size, float *G){
	int index = blockIdx.x*(blockDim.x+2*DEV_PADDING) + (threadIdx.x+DEV_PADDING) + DEV_PADDING*DEV_PIXELS + 2*DEV_PADDING*DEV_PADDING;
	int row_size = DEV_PIXELS + 2*DEV_PADDING;
	//safety-check if
	if(index < padded_size){
		//shared memory
		extern __shared__ float shared_memory[];
		//each thread handles its column
		for(int i=0; i<DEV_PATCH_SIZE; i++){
			shared_memory[(threadIdx.x + DEV_PADDING) + i*row_size] = image[(threadIdx.x+DEV_PADDING) + i*row_size];
		}
		//thread #0 also handles the side paddings
		if(threadIdx.x == 0){
			for(int row=0; row<DEV_PADDING; row++){
				for(int col=0; col<DEV_PATCH_SIZE; col++){
					shared_memory[row + col*row_size] = -1;
				}
			}
			for(int row=(DEV_PADDING+DEV_PIXELS); row<row_size; row++){
				for(int col=0; col<DEV_PATCH_SIZE; col++){
					shared_memory[row + col*row_size] = -1;
				}
			}
		}
		__syncthreads();
		//creating i's patch
		float patch_i[DEV_PATCH_SIZE*DEV_PATCH_SIZE];
		for(int it1=0; it1<DEV_PATCH_SIZE; it1++){
			for(int it2=0; it2<DEV_PATCH_SIZE; it2++){
				patch_i[it1*DEV_PATCH_SIZE + it2] = image[index + (it1-DEV_PADDING)*row_size + it2 - DEV_PADDING];
			}
		}
		filtered_image[index] = 0;
		float weight;
		float Z = 0;
		for(int it1=DEV_PADDING; it1<(DEV_PIXELS+DEV_PADDING); it1++){
			for(int it2=DEV_PADDING; it2<(DEV_PIXELS+DEV_PADDING); it2++){	
				float comp_value = 0;
				compare_patches(&comp_value, patch_i, it2, G, shared_memory);
				weight = (float)(exp(-comp_value/(DEV_FILTER_SIGMA*DEV_FILTER_SIGMA)));
				filtered_image[index] += weight * shared_memory[DEV_PADDING*row_size + it2];
				Z += weight;
			}
			__syncthreads();
			//alter the shared memory, slide everything one row up
			for(int i=0; i<DEV_PATCH_SIZE-1; i++){
				shared_memory[(threadIdx.x+DEV_PADDING) + i*row_size] = shared_memory[(threadIdx.x+DEV_PADDING) + (i+1)*row_size];
			}
			int row_offset = (it1+1-DEV_PADDING)*row_size;
			//insert the new row in the shared_memory
			shared_memory[(threadIdx.x+DEV_PADDING) + (DEV_PATCH_SIZE-1)*row_size] = image[row_offset + (threadIdx.x+DEV_PADDING) + (DEV_PATCH_SIZE-1)*row_size];
			__syncthreads();
		}
		filtered_image[index] = filtered_image[index] / Z; 
	}
}

//! Compares patch_i with the patch of pixel j
__device__ void compare_patches(float *comp_value, float *patch_i, int j, float *G, float *shared_memory){
	int offset = DEV_PADDING*(DEV_PIXELS + 2*DEV_PADDING);
	j += offset;
    for(int it1=0; it1<DEV_PATCH_SIZE; it1++){
        for(int it2=0; it2<DEV_PATCH_SIZE; it2++){
			int first_index = it1*DEV_PATCH_SIZE+it2;
			int second_index = j+(it1-DEV_PADDING)*(DEV_PIXELS+2*DEV_PADDING) + it2 - DEV_PADDING;
            // patch/shared_memory[x] == -1 means it's the added padding
			if(patch_i[first_index] != (float)-1 && shared_memory[second_index] != (float)-1){
                float diff = patch_i[first_index] - shared_memory[second_index];
                *comp_value += G[first_index]*(diff*diff);
            }
        }
    }
}
